#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

// Function to measure execution time using gettimeofday()
double measureExecutionTimeGettimeofday(struct timeval start, struct timeval end) {
    return (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) * 1e-6;
}

__global__ void sumKernel(int *args, int n, int *blockSums) {
    extern __shared__ int sharedData[];

    int tid = threadIdx.x;
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Charger les données en mémoire partagée
    sharedData[tid] = (globalIdx < n) ? args[globalIdx] : 0;
    __syncthreads();

    if (tid == 0) 
        for (int y = 1; y < n; y++) 
            sharedData[tid] += sharedData[y];

    // Seul le thread 0 stocke le résultat du bloc
    if (tid == 0) {
        blockSums[blockIdx.x] = sharedData[0];
    }
}

// Kernel pour additionner les sommes des blocs
__global__ void finalSumKernel(int *blockSums, int numBlocks, int *result) {

    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
 
    if (globalIdx == 0) {
        for (int i = 1; i < numBlocks; i++)
            blockSums[0] += blockSums[i];
        *result = blockSums[0];
    }
}

int main(int argc, char *argv[]) {
    if (argc < 5) {
        printf("Usage: %s size grid_size block_size num1 num2 [...]\n", argv[0]);
        return 1;
    }
    
    int nbArgs = atoi(argv[1]);
    int gridSize = atoi(argv[2]);
    int blockSize = atoi(argv[3]);
    
    int *h_args = (int *)malloc(nbArgs * sizeof(int));
    int h_result = 0;
    
    int *d_args, *d_blockSums, *d_result;
    
    for (int i = 0; i < nbArgs; i++) {
        h_args[i] = atoi(argv[i + 4]);
    }
    
    hipMalloc((void **)&d_args, nbArgs * sizeof(int));
    hipMalloc((void **)&d_blockSums, gridSize * sizeof(int)); // Stockage des sommes partielles
    hipMalloc((void **)&d_result, sizeof(int));
    
    hipMemcpy(d_args, h_args, nbArgs * sizeof(int), hipMemcpyHostToDevice);
    
    
    struct timeval add_start_tv;
    struct timeval add_end_tv;
    gettimeofday(&add_start_tv, NULL);
    //Somme partielle dans chaque bloc
    sumKernel<<<gridSize, blockSize, blockSize * sizeof(int)>>>(d_args, nbArgs, d_blockSums);
    
    //Somme globale des blocs
    finalSumKernel<<<1, gridSize, gridSize * sizeof(int)>>>(d_blockSums, gridSize, d_result);
    
    gettimeofday(&add_end_tv, NULL);
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Sum: %d\n", h_result);
    
    hipFree(d_args);
    hipFree(d_blockSums);
    hipFree(d_result);
    free(h_args);
    
    
    double sum_time_gettimeofday = measureExecutionTimeGettimeofday(add_start_tv, add_end_tv);
    
    
    printf("Sum_time: %f\n", sum_time_gettimeofday);
    
    return sum_time_gettimeofday;
}
